#include "hip/hip_runtime.h"
#include "quant.h"

template <typename T> __device__ __half int_to_half(T value) {
  return __int2half_rn(static_cast<int>(value));
}

__global__ void sym_quantize_f16_i4_kernel(const half *__restrict__ x,
                                           const half *__restrict__ scale,
                                           uint32_t rows, uint32_t colsSrc,
                                           uint32_t colsDst,
                                           Int4Storage *__restrict__ q) {
  uint32_t row = threadIdx.y + blockIdx.y * blockDim.y;
  uint32_t colDst = threadIdx.x + blockIdx.x * blockDim.x;
  if (row >= rows || colDst * kElementsPerVector >= colsSrc) {
    return;
  }
  Int4Storage storage;
  memset(&storage, 0, sizeof(storage));
  uint32_t id = colDst * kElementsPerVector + row * colsSrc;
#pragma unroll
  for (int i = 0; i < kElementsPerVector; ++i) {
    bool safe = (colDst * kElementsPerVector + i) < colsSrc;
    if (safe) {
      half data = __hdiv(x[id + i], scale[row]);
      int qval = clamp(__half2int_rn(data), qmin, qmax);
      Int4Subbyte{reinterpret_cast<cutlass::int4b_t *>(&storage), i}.set(qval);
    }
  }

  q[colDst + row * colsDst] = storage;
}

void sym_quant_host(const half *x, const half *scale, uint32_t rows,
                    uint32_t colsSrc, uint32_t colsDst, Int4Storage *q) {

  dim3 block{std::min<uint32_t>(colsDst, 32), std::min<uint32_t>(rows, 16)};
  dim3 grid{cdiv(colsDst, block.x), cdiv(rows, block.y)};
  sym_quantize_f16_i4_kernel<<<grid, block>>>(x, scale, rows, colsSrc, colsDst, q);
}

__global__ void
sym_dequantize_i32_f16_kernel(const int32_t *__restrict__ q,
                              const half *__restrict__ scale_row,
                              const half *__restrict__ scale_col, uint32_t rows,
                              uint32_t cols, half *__restrict__ x) {
  uint32_t row = threadIdx.y + blockIdx.y * blockDim.y;
  uint32_t col = threadIdx.x + blockIdx.x * blockDim.x;

  if (col >= cols || row >= rows) {
    return;
  }

  half xElement = int_to_half(q[col + row * cols]);
  x[col + row * cols] = scale_row[row] * scale_col[col] * xElement;
}

void sym_dequant_host(const int32_t *q, const half *scale_row,
                      const half *scale_col, uint32_t rows, uint32_t cols,
                      half *x) {
  dim3 block{std::min<uint32_t>(cols, 16), std::min<uint32_t>(rows, 16)};
  dim3 grid{cdiv(cols, block.x), cdiv(rows, block.y)};
  sym_dequantize_i32_f16_kernel<<<grid, block>>>(q, scale_row, scale_col, rows, cols, x);
}

__global__ void
sym_dequantize_col_only_i4_f16_kernel(const int8_t *__restrict__ q,
                                       const half *__restrict__ scale_col, 
                                       uint32_t rowsSrc, uint32_t rowsDst,
                                       uint32_t cols, half *__restrict__ x) {
  uint32_t rowSrc = threadIdx.y + blockIdx.y * blockDim.y;
  uint32_t col = threadIdx.x + blockIdx.x * blockDim.x;

  if (col >= cols || rowSrc * kElementsPerVector >= rowsDst) {
    return;
  }

  uint32_t id = col + rowSrc * cols;
  uint32_t src_qval = q[id];
  uint32_t qval = 0;

#pragma unroll
  for (int i = 0; i < kElementsPerVector; ++i) {
    bool safe = (rowSrc * kElementsPerVector + i) < rowsDst;
    if (safe) {
      // load the 4bit value
      qval = src_qval & 0xf;
      src_qval >>= 4;
      x[col + (rowSrc * kElementsPerVector + i) * cols] = scale_col[col] * int_to_half(qval);
    }
  }
}

void sym_dequant_col_only_host(const int8_t *q, const half *scale_col, uint32_t rowsSrc, uint32_t rowsDst,
                               uint32_t cols, half *x) {
  dim3 block{std::min<uint32_t>(cols, 16), std::min<uint32_t>(rowsSrc, 16)};
  dim3 grid{cdiv(cols, block.x), cdiv(rowsSrc, block.y)};
  sym_dequantize_col_only_i4_f16_kernel<<<grid, block>>>(q, scale_col, rowsSrc, rowsDst, cols, x);
}

__global__ void
sym_dequantize_row_only_i4_f16_kernel(const int8_t *__restrict__ q,
                                       const half *__restrict__ scale_row, 
                                       uint32_t rows, uint32_t colsSrc,
                                       uint32_t colsDst, half *__restrict__ x) {
  uint32_t row = threadIdx.y + blockIdx.y * blockDim.y;
  uint32_t colSrc = threadIdx.x + blockIdx.x * blockDim.x;

  if (row >= rows || colSrc * kElementsPerVector >= colsDst) {
    return;
  }

  uint32_t id = colSrc + row * colsSrc;
  uint32_t src_qval = q[id];
  uint32_t qval = 0;

#pragma unroll
  for (int i = 0; i < kElementsPerVector; ++i) {
    bool safe = (colSrc * kElementsPerVector + i) < colsDst;
    if (safe) {
      // load the 4bit value
      qval = src_qval & 0xf;
      src_qval >>= 4;
      x[colSrc * kElementsPerVector + i + row * colsDst] = scale_row[row] * int_to_half(qval);
    }
  }
}

void sym_dequant_row_only_host(const int8_t *q, const half *scale_row, uint32_t rows, uint32_t colsSrc,
                               uint32_t colsDst, half *x) {
  dim3 block{std::min<uint32_t>(colsSrc, 16), std::min<uint32_t>(rows, 16)};
  dim3 grid{cdiv(colsSrc, block.x), cdiv(rows, block.y)};
  sym_dequantize_row_only_i4_f16_kernel<<<grid, block>>>(q, scale_row, rows, colsSrc, colsDst, x);
}

__global__ void
sym_dequantize_quantize_i4_f16_i4_kernel(const int8_t *__restrict__ q_in,
                                         int8_t *__restrict__ q_out,
                                         const half *__restrict__ scale_row,
                                         const half *__restrict__ scale_col, 
                                         uint32_t rowsSrc, uint32_t rowsDst, // rowsSrc is small, rowsDst is big
                                         uint32_t colsSrc, uint32_t colsDst // colsSrc is small, colsDst is big
) {
  uint32_t rowSrc = threadIdx.y + blockIdx.y * blockDim.y;
  uint32_t colSrc = threadIdx.x + blockIdx.x * blockDim.x;

  __shared__ half buffer[64][64];

  if (colSrc * kElementsPerVector >= colsDst || rowSrc * kElementsPerVector >= rowsDst) {
    return;
  }

  // first, row-wise dequantize, and move to shared memory
  uint32_t id = colSrc + rowSrc * colsSrc;
  uint32_t src_qval = q_in[id];
  uint32_t qval = 0;

#pragma unroll
  for (int i = 0; i < kElementsPerVector; ++i) {
    bool safe = (colSrc * kElementsPerVector + i) < colsDst;
    if (safe) {
      // load the 4bit value
      qval = src_qval & 0xf;
      src_qval >>= 4;
      buffer[rowSrc][colSrc * kElementsPerVector + i] = scale_row[rowSrc] * int_to_half(qval);
    }
  }

  __syncthreads();

  // second, col-wise quantize
  Int4Storage storage;
  memset(&storage, 0, sizeof(storage));

#pragma unroll
  for (int i = 0; i < kElementsPerVector; ++i) {
    bool safe = (rowSrc * kElementsPerVector + i) < rowsDst;
    if (safe) {
      half data = buffer[rowSrc * kElementsPerVector + i][colSrc];
      int qval = clamp(__half2int_rn(data), qmin, qmax);
      Int4Subbyte{reinterpret_cast<cutlass::int4b_t *>(&storage), i}.set(qval);
    }
  }

  // third, transpose
  // original position: colSrc + rowSrc * colsDst
  // transposed position: rowSrc + colSrc * rowsDst
  q_out[rowSrc + colSrc * rowsDst] = storage;
}

void sym_dequantize_quantize_host(const int8_t *q_in, int8_t *q_out,
                                  const half *scale_row, const half *scale_col, 
                                  uint32_t rowsSrc, uint32_t rowsDst, uint32_t colsSrc, uint32_t colsDst) {
  dim3 block{std::min<uint32_t>(colsDst, 32), std::min<uint32_t>(rowsDst, 32)};
  dim3 grid{cdiv(colsDst, block.x), cdiv(rowsDst, block.y)};
  sym_dequantize_quantize_i4_f16_i4_kernel<<<grid, block>>>(q_in, q_out, scale_row, scale_col, rowsSrc, rowsDst, colsSrc, colsDst);
}
